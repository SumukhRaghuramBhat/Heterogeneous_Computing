
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#define N 1000000        //Job size = 1M
#define M 128 		// Varying Thread block size- 128, 256, 512, 1024

//add kernel
__global__ void add(int *a, int *b, int *c, int k)  
{
	int index = threadIdx.x+ blockIdx.x * blockDim.x;
	if (index<k)
		c[index] = a[index] + b[index];
}

//Random number generator function
void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%N;
	}
}

int main(void)
{

	int *a, *b, *c; 
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	//time start and stop
	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//Allocate CPU memory
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipEventRecord( start, 0 );
	
	//Copy CPU memory to GPU memory	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	
	//Call the add kernel
	add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c,N); 
	
	printf("GPU Execution Time = %f\n",time);
	
	// Copy from device to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("Execution Time = %f\n",time);
	
	//Cleanup
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;

}



