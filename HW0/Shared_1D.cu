
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

#define N   10000000 	//job size = 1K, 10K, 100K, 1M and 10M
#define M   128 	//Threads per block =128
#define R   16 		//radius = 2,4,8,16

// CUDA API error checking macro
static void handleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) 
{
	__shared__ int temp[M + 2 * R];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + R;
	temp[lindex] = in[gindex]; // Read input elements into shared memory
	
	if (threadIdx.x < R) 
	{
	temp[lindex - R] = in[gindex - R];
	temp[lindex + M] = in[gindex + M];
	}
	
	// Synchronize (ensure all the data is available)
	__syncthreads();
	int result = 0;
	
	// Apply the stencil
	for (int offset = -R ; offset <= R ; offset++)
	{
		result += temp[lindex + offset];
	}
	
	// Store the result
	out[gindex] = result;
}	

int main()
{
	unsigned int i;
	int h_in[N + 2 * R], h_out[N];
	int *d_in, *d_out;

	//time start and stop
	hipEvent_t start, stop;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	for( i = 0; i < (N + 2*R); ++i )
	h_in[i] = 1; 

	// Allocate device memory
	cudaCheck( hipMalloc( &d_in, (N + 2*R) * sizeof(int)) );
	cudaCheck( hipMalloc( &d_out, N * sizeof(int)) );

	//copy fro CPU to GPU memory
	cudaCheck( hipMemcpy( d_in, h_in, (N + 2*R) * sizeof(int), hipMemcpyHostToDevice) );
	hipEventRecord( start, 0 );

	//Call stencil kernel
	stencil_1d<<< (N + M - 1)/M, M >>> (d_in, d_out);

	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("GPU Execution Time = %f\n",time);

	//copy from device to host
	cudaCheck( hipMemcpy( h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost) );

	// Cleanup
	hipFree(d_in);
	hipFree(d_out);

  return 0;
}

