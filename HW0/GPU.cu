
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

#define N 1000 		//Job size = 1K, 10K, 100K, 1M and 10M

//add kernel
__global__ void add(int *a, int *b, int *c)  
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

//function to generate random numbers 
void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%N;
	}
}

int main(void)
{

	int *a, *b, *c; 	// host copies of a, b, c
	int *d_a, *d_b, *d_c;		// device copies of a, b, c
	int size = N * sizeof(int);
	
	//time start and stop
	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start); 
	hipEventCreate(&stop);

	//Allocate device memory
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//Allocate CPU memory 
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	hipEventRecord( start, 0 );
	
	//Copy CPU memory to GPU memory
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	
	// Launch add() kernel on GPU with N blocks
	add<<<1,N>>>(d_a, d_b, d_c); //N Threads and 1 Thread Block 

	//Copy from device to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );

	printf("GPU Execution Time = %f\n",time);
	
	//Cleanup
	free(a); 
	free(b);
	free(c);
	hipFree(d_a); 
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}



