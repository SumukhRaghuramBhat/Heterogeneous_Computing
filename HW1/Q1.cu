
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define WA 10000	// Matrix A width
#define HA 10000	// Matrix A height
#define WB 10000	// Matrix B width
#define HB WA		// Matrix B height
#define WC WB		// Matrix C width 
#define HC HA		// Matrix C height

#define N 100
#define M 100
#define BLOCK_SIZE 16


// Allocates a matrix with random float entries
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}


// Code running on GPU //

__global__ void
matrixMul_naive( float* C, float* A, float* B, int wA, int wB)
{
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int i = by * blockDim.y + ty;
  int j = bx * blockDim.x + tx;

  float accu = 0.0;

  for(int k=0; k<wA; k++){
    accu = accu + A[ i * wA + k ] * B[ k * wB + j ];
  }
  C[ i * wB + j ] = accu;
}

int main(){
    hipEvent_t start, stop;    
    float msecTotal;
   
    hipEventCreate(&start);

    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    float* h_C = (float*) malloc(mem_size_C);
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, d_B, mem_size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N)/dimBlock.x, (M)/dimBlock.y);

    hipEventRecord(start, NULL); 


    // execute the kernel
    matrixMul_naive<<< dimGrid, dimBlock >>>(d_C, d_A, d_B,WA,WB);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
   
    printf("GPU Execution time: %f (ms) \n", msecTotal);

return 0;
    
}



