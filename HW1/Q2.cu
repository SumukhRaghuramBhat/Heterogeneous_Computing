#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#define WA 10000	// Matrix A width
#define HA 10000	// Matrix A height
#define WB 10000	// Matrix B width
#define HB WA		// Matrix B height
#define WC WB		// Matrix C width 
#define HC HA		// Matrix C height

#define N 100
#define M 100
#define BLOCK_SIZE 16

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void
matrixMul_coalescing( float* C, float* A, float* B, int wA, int wB)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int aBegin = wA * BLOCK_SIZE * by;

    int aEnd   = aBegin + wA - 1;

    int aStep  = BLOCK_SIZE;

    int bBegin = BLOCK_SIZE * bx;

    int bStep  = BLOCK_SIZE * wB;
    float Csub = 0;

    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {
        As(ty, tx) = A[a + wA * ty + tx];
        Bs(tx, ty) = B[b + wB * ty + tx];

        __syncthreads();

             for (int k = 0; k < BLOCK_SIZE; ++k)
	  Csub += AS(ty, k) * BS(tx, k);

                __syncthreads();
    }

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

/* Codes running on GPU */

int main(){
    hipEvent_t start, stop; 
    float msecTotal;

    hipEventCreate(&start);

    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    float *transpB;
    transpB = (float *) malloc(mem_size_B);

    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;

    hipMalloc((void**) &d_C, mem_size_C);
    float* h_C = (float*) malloc(mem_size_C);
   
    for (int i = 0; i < M; i++)
		for (int j = 0; j < M; j++)
			*(transpB + i * M + j) = *(h_B + j * M + i);

    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, transpB, mem_size_B, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N)/dimBlock.x, (M)/dimBlock.y);

    hipEventRecord(start, NULL); 

    // execute the kernel
    matrixMul_coalescing<<< dimGrid, dimBlock >>>(d_C, d_A, d_B,WA,WB);

    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
      
    printf("GPU Execution time: %f (ms) \n", msecTotal);

return 0;
    
}



